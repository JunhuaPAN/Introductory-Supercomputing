#include <stdio.h>
#include <string.h>
#include <mpi.h>
#include <hip/hip_runtime.h>
int devCount;
int myid;
int ihavecuda;
int deviceselector=0;


int main(int argc, char* argv[]) {
	int myrank;
	char hostname[MPI_MAX_PROCESSOR_NAME];
	int resultlen;
	/* Initialize the MPI library */
	MPI_Init(&argc, &argv);
	/* Determine unique id of the calling process of all processes participating
	   in this MPI program. This id is usually called MPI rank. */
	MPI_Comm_rank(MPI_COMM_WORLD, &myrank);
	/* Get hostname where the MPI-tasks are running*/
	MPI_Get_processor_name(hostname, &resultlen);
	/*CUDA segment of work invoked on each MPI-task as host*/
	hipGetDeviceCount(&devCount);
	if (devCount == 0) {
		printf("[Hostname=%s | rank=%d]: \nDevcount %4d NONE\n",hostname,myrank,devCount);
		ihavecuda=0;
	}
	else{
		ihavecuda=1;
		if (devCount >= 1){
			printf("[Hostname=%s | rank=%d]: \nDevcount %4d NONE\n",hostname,myrank,devCount);
			for (int i = 0; i < devCount; ++i)
			{
				hipDeviceProp_t devProp;
				hipGetDeviceProperties(&devProp, i);
				printf("[rank=%d]: devprop name %s i=(%d) \n ",myrank, devProp.name, i);
			}
		}
	}

	/* Finalize the MPI library to free resources acquired by it. */
	MPI_Finalize();
	return 0;


}


